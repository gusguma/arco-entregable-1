///////////////////////////////////////////////////////////////////////////
/// PROGRAMACIÓN EN CUDA C/C++
/// Práctica:	ENTREGABLE 1 : Temporización GPU
/// Autor:		Angel Sierra Gomez, Gustavo Gutierrez Martin
/// Fecha:		Noviembre 2022
///////////////////////////////////////////////////////////////////////////

/// dependencias ///
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

/// constantes ///
#define MB (1<<20) /// MiB = 2^20
#define ROWS 7
#define COLUMNS 24

/// numero de CUDA cores
int getCudaCores(hipDeviceProp_t deviceProperties);
/// realiza la suma de los arrays en el device
__global__ void transfer(const int *dev_vector, int *dev_result);

int main() {
    int deviceCount;
    int *hst_vector,*hst_result;
    int *dev_vector,*dev_result;
    dim3 blocks(1);
    dim3 threads(ROWS, COLUMNS);
    /// declaracion de eventos
    hipEvent_t start;
    hipEvent_t stop;

    /// buscando dispositivos
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        /// mostramos el error si no se encuentra un dispositivo
        printf("¡No se ha encontrado un dispositivo CUDA!\n");
        printf("<pulsa [INTRO] para finalizar>");
        getchar();
        return 1;
    } else {
        printf("Se han encontrado %d dispositivos CUDA:\n", deviceCount);
        for (int deviceID = 0; deviceID < deviceCount; deviceID++) {
            ///obtenemos las propiedades del dispositivo CUDA
            hipDeviceProp_t deviceProp{};
            hipGetDeviceProperties(&deviceProp, deviceID);
            int SM = deviceProp.multiProcessorCount;
            int cudaCores = getCudaCores(deviceProp);
            printf("***************************************************\n");
            printf("DEVICE %d: %s\n", deviceID, deviceProp.name);
            printf("***************************************************\n");
            printf("- Capacidad de Computo            \t: %d.%d\n", deviceProp.major, deviceProp.minor);
            printf("- No. de MultiProcesadores        \t: %d \n", SM);
            printf("- No. de CUDA Cores (%dx%d)       \t: %d \n", cudaCores, SM, cudaCores * SM);
            printf("- Memoria Global (total)          \t: %zu MiB\n", deviceProp.totalGlobalMem / MB);
            printf("- No. maximo de Hilos (por bloque)\t: %d\n", deviceProp.maxThreadsPerBlock);
            printf("***************************************************\n");
        }
    }
    /// reserva del espacio de memoria en el host
    hst_vector = (int*)malloc( ROWS * COLUMNS * sizeof(int));
    hst_result = (int*)malloc( ROWS * COLUMNS * sizeof(int));
    /// reserva del espacio de memoria en el device
    hipMalloc( (void**)&dev_vector, ROWS * COLUMNS * sizeof(int) );
    hipMalloc( (void**)&dev_result, ROWS * COLUMNS * sizeof(int) );
    /// cargamos los datos iniciales en el host
    srand ( (int)time(nullptr) );
    for (int i = 0; i < ROWS; i++)  {
        int number = (int) rand() % 9 + 1;
        for (int j=0; j < COLUMNS; j++) {
            /// inicializamos hst_vector1 con numeros aleatorios entre 0 y 1
            hst_vector[i * COLUMNS + j] = number;
            /// inicializamos hst_vector2 con ceros
            hst_result[i * COLUMNS + j] = 0;
        }
    }
    /// creacion de eventos
    hipEventCreate(&start);
    hipEventCreate(&stop);
    /// transfiere datos de host a device
    hipMemcpy(dev_vector,hst_vector, ROWS * COLUMNS * sizeof(int),hipMemcpyHostToDevice);
    /// mostramos los datos con los que llamamos al device
    printf("Lanzamiento de: %d bloque y %d hilos \n", 1, threads.x * threads.y);
    printf("> Eje X: %d \n", threads.x);
    printf("> Eje Y: %d \n", threads.y);
    printf("***************************************************\n");
    /// marca de inicio
    hipEventRecord(start,nullptr);
    /// sumamos los items
    transfer<<< blocks, threads >>>(dev_vector, dev_result);
    /// marca de final
    hipEventRecord(stop,nullptr);
    /// sincronizacion GPU-CPU
    hipEventSynchronize(stop);
    /// cálculo del tiempo en milisegundos
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime,start,stop);
    /// impresion de resultados
    printf("> Tiempo de ejecucion: %f ms\n",elapsedTime);
    printf("***************************************************\n");
    /// transferimos los datos del device al host
    hipMemcpy(hst_result, dev_result, ROWS * COLUMNS * sizeof(int), hipMemcpyDeviceToHost);
    /// muestra por pantalla los datos del host
    printf("MATRIZ ORIGINAL:\n");
    for (int i = 0; i < ROWS; i++)  {
        for (int j = 0; j < COLUMNS; j++) {
            printf("%d ", hst_vector[j + i * COLUMNS]);
        }
        printf("\n");
    }
    printf("\n");
    printf("MATRIZ FINAL:\n");
    for (int i = 0; i < ROWS; i++)  {
        for (int j = 0; j < COLUMNS; j++) {
            printf("%d ", hst_result[j + i * COLUMNS]);
        }
        printf("\n");
    }
    printf("\n");

    /// función que muestra por pantalla la salida del programa
    time_t fecha;
    time(&fecha);
    printf("***************************************************\n");
    printf("Programa ejecutado el: %s", ctime(&fecha));
    printf("***************************************************\n");
    /// capturamos un INTRO para que no se cierre la consola de MSVS
    printf("<pulsa [INTRO] para finalizar>");
    getchar();

    /// liberacion de recursos
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(dev_vector);
    hipFree(dev_result);
    return 0;
}

int getCudaCores(hipDeviceProp_t deviceProperties) {
    int cudaCores = 0;
    int major = deviceProperties.major;
    if (major == 1) {
        /// TESLA
        cudaCores = 8;
    } else if (major == 2) {
        /// FERMI
        if (deviceProperties.minor == 0) {
            cudaCores = 32;
        } else {
            cudaCores = 48;
        }
    } else if (major == 3) {
        /// KEPLER
        cudaCores = 192;
    } else if (major == 5) {
        /// MAXWELL
        cudaCores = 128;
    } else if (major == 6 || major == 7 || major == 8) {
        /// PASCAL, VOLTA (7.0), TURING (7.5), AMPERE
        cudaCores = 64;
    } else {
        /// ARQUITECTURA DESCONOCIDA
        cudaCores = 0;
        printf("¡Dispositivo desconocido!\n");
    }
    return cudaCores;
}

__global__ void transfer(const int *dev_vector, int *dev_result) {
    /// identificador del hilo
    unsigned int threadX = threadIdx.y;
    unsigned int threadY = threadIdx.x;
    /// calculamos el ID  hilo
    unsigned int myID = threadY + threadX * blockDim.x;
    /// calculamos la fila donde se encuentra la posicion
    int row = (int) myID / COLUMNS;
    /// calculamos si la posicion
    if (row < (ROWS - 1)) {
        dev_result[myID + COLUMNS] = dev_vector[myID];
    } else {
        dev_result[myID - (COLUMNS * (ROWS - 1))] = dev_vector[myID];
    }
}
