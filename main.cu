///////////////////////////////////////////////////////////////////////////
/// PROGRAMACIÓN EN CUDA C/C++
/// Práctica:	ENTREGABLE 1 : Temporización GPU
/// Autor:		Angel Sierra Gomez, Gustavo Gutierrez Martin
/// Fecha:		Noviembre 2022
///////////////////////////////////////////////////////////////////////////

/// dependencias ///
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>

/// constantes ///
#define MB (1<<20) /// MiB = 2^20
#define ROWS 7
#define COLUMNS 24





int main() {
    std::cout << "Hello, World!" << std::endl;
    return 0;
}
